#include "globalFunctions.cuh"
#include "bcsFunctions.cuh"
#include "init.cuh"
#include "lbm.cuh"
#include "bcs.cuh"
#include "../helpers/hostFunctions.cuh"
#if defined(D_FIELDS)
#include "../helpers/derivedFields.cuh"
#endif 

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Error: Usage: " << argv[0] << " <velocity set> <ID>\n";
        return 1;
    }
    const std::string VELOCITY_SET = argv[1];
    const std::string SIM_ID       = argv[2];
    const std::string SIM_DIR = createSimulationDirectory(VELOCITY_SET, SIM_ID);

    //#define BENCHMARK
    setDevice();
    
    constexpr dim3 block(32u, 2u, 2u); 
    constexpr dim3 blockX(16u, 16u, 1u);
    constexpr dim3 blockY(16u, 16u, 1u);
    constexpr dim3 blockZ(16u, 16u, 1u);

    constexpr dim3 grid(divUp(NX, block.x),
                        divUp(NY, block.y),
                        divUp(NZ, block.z));

    constexpr dim3 gridX(divUp(NY, blockX.x),
                         divUp(NZ, blockX.y),
                         1u);

    constexpr dim3 gridY(divUp(NX, blockY.x),
                         divUp(NZ, blockY.y),
                         1u);

    constexpr dim3 gridZ(divUp(NX, blockZ.x),
                         divUp(NY, blockZ.y),
                         1u);

    constexpr size_t dynamic = 0;

    hipStream_t queue{};
    checkCudaErrors(hipStreamCreate(&queue));

    // =========================== INITIALIZATION =========================== //

        setFields<<<grid, block, dynamic, queue>>>(lbm);
        setOilJet<<<grid, block, dynamic, queue>>>(lbm);
        setWaterJet<<<grid, block, dynamic, queue>>>(lbm);
        setDistros<<<grid, block, dynamic, queue>>>(lbm);
    
    // ===================================================================== //

    const auto START_TIME = std::chrono::high_resolution_clock::now();
    for (int STEP = 0; STEP <= NSTEPS; ++STEP) {

        // ======================== LATTICE BOLTZMANN RELATED ======================== //

            computePhase<<<grid, block, dynamic, queue>>>(lbm);
            forceStreamCollide<<<grid, block, dynamic, queue>>>(lbm);

        // ========================================================================== //


        // ============================== BOUNDARY CONDITIONS ============================== //
        
            applyOilInflow<<<gridZ, blockZ, dynamic, queue>>>(lbm);
            applyWaterInflow<<<gridY, blockY, dynamic, queue>>>(lbm);
            applyOutflowZ<<<gridZ, blockZ, dynamic, queue>>>(lbm);
            applyOutflowY<<<gridY, blockY, dynamic, queue>>>(lbm);
            periodicX   <<<gridX, blockX, dynamic, queue>>>(lbm);
            //periodicY   <<<gridY, blockY, dynamic, queue>>>(lbm);

        // ================================================================================= //

        #if defined(D_FIELDS)
        computeDerived<<<grid, block, dynamic, queue>>>(lbm, dfields);
        #endif 

        #if !defined(BENCHMARK)

        checkCudaErrors(hipDeviceSynchronize());

        if (STEP % MACRO_SAVE == 0) {

            //copyAndSaveToBinary(lbm.rho, PLANE, SIM_DIR, SIM_ID, STEP, "rho");
            copyAndSaveToBinary(lbm.phi, PLANE, SIM_DIR, SIM_ID, STEP, "phi");
            copyAndSaveToBinary(lbm.uz,  PLANE, SIM_DIR, SIM_ID, STEP, "uz");
            #if defined(D_FIELDS)
            copyAndSaveToBinary(dfields.vorticity_mag, PLANE, SIM_DIR, SIM_ID, STEP, "vorticity_mag");
            copyAndSaveToBinary(dfields.velocity_mag,  PLANE, SIM_DIR, SIM_ID, STEP, "velocity_mag");
            #endif 
            std::cout << "Step " << STEP << ": bins in " << SIM_DIR << "\n";

        }

        #endif
    }

    const auto END_TIME = std::chrono::high_resolution_clock::now();
    checkCudaErrorsOutline(hipStreamDestroy(queue));

    hipFree(lbm.f);
    hipFree(lbm.g);
    hipFree(lbm.phi);
    hipFree(lbm.rho);
    hipFree(lbm.normx);
    hipFree(lbm.normy);
    hipFree(lbm.normz);
    hipFree(lbm.ux);
    hipFree(lbm.uy);
    hipFree(lbm.uz);
    hipFree(lbm.pxx);
    hipFree(lbm.pyy);
    hipFree(lbm.pzz);
    hipFree(lbm.pxy);
    hipFree(lbm.pxz);
    hipFree(lbm.pyz);

    #if defined(D_FIELDS)
    hipFree(dfields.vorticity_mag);
    hipFree(dfields.velocity_mag);
    #endif 

    const std::chrono::duration<double> ELAPSED_TIME = END_TIME - START_TIME;
    const uint64_t TOTAL_CELLS = static_cast<uint64_t>(NX) * NY * NZ * static_cast<uint64_t>(NSTEPS ? NSTEPS : 1);
    const double   MLUPS       = static_cast<double>(TOTAL_CELLS) / (ELAPSED_TIME.count() * 1e6);

    std::cout << "\n// =============================================== //\n";
    std::cout << "     Total execution time    : " << ELAPSED_TIME.count() << " s\n";
    std::cout << "     Performance             : " << MLUPS << " MLUPS\n";
    std::cout << "// =============================================== //\n\n";

    generateSimulationInfoFile(SIM_DIR, SIM_ID, VELOCITY_SET, MLUPS);
    getLastCudaErrorOutline("Final sync");

    return 0;
}
